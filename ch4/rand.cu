#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hip/driver_types.h>
#include <stdio.h>

#pragma comment(lib, "cudart.lib")
#pragma comment(lib, "curand.lib")

int main() {
  int N = 1024;
  hiprandGenerator_t gen;
  float *p_d, *p_h;

  p_h = (float *)malloc(N * sizeof(float));
  hipMalloc((void **)&p_d, N * sizeof(float));

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 9999ULL);
  hiprandGenerateUniform(gen, p_d, N);

  hipMemcpy(p_h, p_d, N * sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("%.4f\n", p_h[i]);
  }

  hiprandDestroyGenerator(gen);
  hipFree(p_d);
  free(p_h);
}
