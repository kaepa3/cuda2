#include <corecrt_malloc.h>
#include <hip/hip_complex.h>
#include <cuda_device_runtime_api.h>
#include <hipfft/hipfft.h>
#include <hip/driver_types.h>
#include <stdio.h>
#pragma comment(lib, "cufft.lib")

int main() {
#define NX 256
#define NY 256
  hipfftHandle plan;
  hipfftComplex *host;
  hipfftComplex *dev;
  host = (hipfftComplex *)malloc(sizeof(hipfftComplex) * NX * NY);
  for (int i = 0; i < NX; i++) {
    for (int j = 0; j < NX; j++) {
      if (j > NX - 50 && j < NX / 2 + 50 && i > NY / 2 - 50 &&
          i < NY / 2 + 50) {
        host[j + i * NX] = make_hipComplex(1.0f, 0.0f);
      } else {
        host[j + i * NX] = make_hipComplex(0.0f, 0.0f);
      }
    }
  }

  hipMalloc((void **)&dev, sizeof(hipfftComplex));
  hipMemcpy(dev, host, sizeof(float) * NX * NY, hipMemcpyHostToDevice);

  hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C);
  hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
  hipMemcpy(host, dev, sizeof(float) * NX * NY, hipMemcpyDeviceToHost);
  hipfftDestroy(plan);
  hipFree(dev);
  free(host);
}
